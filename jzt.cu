#include "hip/hip_runtime.h"
extern "C" {
	#include "lua.h"
	#include "lualib.h"
	#include "lauxlib.h"
}

#include "luaT.h"
#include "THC.h"

#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

#include <stdio.h>
#include <assert.h>
#include "hipblas.h"

#define TB 128

/* operations */
struct opPlus {
	static const float base_value = 0.0;
	__device__ float operator()(float x, float y)
	{
		return x + y;
	}
};

struct opMinus {
	static const float base_value = 0.0;
	__device__ float operator()(float x, float y)
	{
		return x - y;
	}
};

struct opMult {
	static const float base_value = 1.0;
	__device__ float operator()(float x, float y)
	{
		return x * y;
	}
};

struct opDiv {
	static const float base_value = 1.0;
	__device__ float operator()(float x, float y)
	{
		return x / y;
	}
};

struct opMax {
	static const float base_value = -2e38;
	__device__ float operator()(float x, float y)
	{
		return fmaxf(x, y);
	}
};

struct opExp {
public:
	__device__ float operator()(float x)
	{
		return exp(x);
	}
};

struct opSigmoid {
public:
	__device__ float operator()(float x)
	{
		return 1 / (1 + exp(-x));
	}
};

struct opSigmoidDeriv {
public:
	__device__ float operator()(float x, float y)
	{
		return x * y * (1 - y);
	}
};

struct opTanh {
public:
	__device__ float operator()(float x)
	{
		return tanh(x);
	}
};

struct opTanhDeriv {
public:
	__device__ float operator()(float x, float y)
	{
		return x * (1 - y * y);
	}
};

struct opCCE {
public:
	__device__ float operator()(float input, float target)
	{
		return target > 0 ? target * log(input) : 0;
	}
};

struct opHuber {
	float threshold;
	opHuber(float threshold_) : threshold(threshold_) {};
	__device__ float operator()(float x, float y) {
		float d = x - y;
		if (-threshold < d && d < threshold) {
			return 0.5 * d * d;
		} else {
			return threshold * (abs(d) - 0.5 * threshold);
		}
	}
};

struct opHuberDeriv {
	float threshold;
	opHuberDeriv(float threshold_) : threshold(threshold_) {};
	__device__ float operator()(float x, float y) {
		float d = x - y;
		if (-threshold < d && d < threshold) {
			return d;
		} else {
			return threshold * signbit(d);
		}
	}
};

struct opShrink {
	float threshold;
	opShrink(float threshold_) : threshold(threshold_) {};
	
	__device__ float operator()(float x) { 
		if (x - threshold > 0) {
			return x - threshold;
		} else if (x + threshold < 0) {
			return x + threshold;
		} else {
			return 0.0;
		}
	}
};

/* Is A in row major format? */
int is_rm(THCudaTensor *A)
{
	return A->stride[1] == 1 || A->nDimension == 1;
}

void checkCudaError(lua_State *L) {
	hipError_t status = hipPeekAtLastError();
	if (status != hipSuccess) {
		luaL_error(L, hipGetErrorString(status));
	}
}

/* res[i] = A[inds[i]] */
__global__ void get_cols(float *A, int A_stride, float *inds, float *res, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len) {
		res[i] = A[i * A_stride + (int)inds[i] - 1];
	}
}

int get_cols(lua_State *L)
{
	THCudaTensor *A = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *inds = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *res = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	int len = THCudaTensor_nElement(inds);
	get_cols<<<(len - 1)  / TB + 1, TB>>>(THCudaTensor_data(A), A->stride[0], THCudaTensor_data(inds), THCudaTensor_data(res), len);
	return 0;
}


/* A[inds[i]] = val */
__global__ void set_cols(float *A, int A_stride, float *inds, float val, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len) {
		A[i * A_stride + (int)inds[i] - 1] = val;
	}
}

int set_cols(lua_State *L)
{
	THCudaTensor *A = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *inds = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	float val = luaL_checknumber(L, 3);
	int len = THCudaTensor_nElement(inds);
	set_cols<<<(len - 1)  / TB + 1, TB>>>(THCudaTensor_data(A), A->stride[0], THCudaTensor_data(inds), val, len);
	return 0;
}


template<class Op>
int transform1(Op op, lua_State *L)
{
	THCudaTensor *A = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *B = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	int lenA = THCudaTensor_nElement(A);
	int lenB = THCudaTensor_nElement(B);

	if (!is_rm(A) || !is_rm(B)) {
		luaL_error(L, "Matrices not in row major order");
	}

	if (lenA != lenB) {
		luaL_error(L, "Size mismatch");
	}

	thrust::device_ptr<float> pA(THCudaTensor_data(A));
	thrust::device_ptr<float> pB(THCudaTensor_data(B));
	thrust::transform(pA, pA + lenA, pB, op);
	return 0;
}

template<class Op>
int transform2(Op op, lua_State *L)
{
	THCudaTensor *A = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *B = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *C = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	int lenA = THCudaTensor_nElement(A);
	int lenB = THCudaTensor_nElement(B);
	int lenC = THCudaTensor_nElement(C);

	if (!is_rm(A) || !is_rm(B) || !is_rm(C)) {
		luaL_error(L, "Matrices not in roj major order");
	}

	if (lenA != lenB || lenA != lenC) {
		luaL_error(L, "Size mismatch");
	}

	thrust::device_ptr<float> pA(THCudaTensor_data(A));
	thrust::device_ptr<float> pB(THCudaTensor_data(B));
	thrust::device_ptr<float> pC(THCudaTensor_data(C));
	thrust::transform(pA, pA + lenA, pB, pC, op);
	return 0;
}

int huber(lua_State *L)
{
	float threshold = luaL_checknumber(L, 4);
	return transform2(opHuber(threshold), L);
}

int huber_deriv(lua_State *L)
{
	float threshold = luaL_checknumber(L, 4);
	return transform2(opHuberDeriv(threshold), L);
}

int shrink(lua_State *L)
{
	float threshold = luaL_checknumber(L, 3);
	return transform1(opShrink(threshold), L);
}

int sigmoid(lua_State *L)
{
	return transform1(opSigmoid(), L);
}

int mult_by_sigmoid_deriv(lua_State *L)
{
	return transform2(opSigmoidDeriv(), L);
}

int tanh(lua_State *L)
{
	return transform1(opTanh(), L);
}

int mult_by_tanh_deriv(lua_State *L)
{
	return transform2(opTanhDeriv(), L);
}

int cce(lua_State *L)
{
	THCudaTensor *C = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");

	transform2(opCCE(), L);
	thrust::device_ptr<float> pC(THCudaTensor_data(C));
	float sum = thrust::reduce(pC, pC + THCudaTensor_nElement(C));

	lua_pushnumber(L, -sum);
	return 1;
}

int _exp(lua_State *L)
{
	return transform1(opExp(), L);
}

/* What a crazy bug!
 *
 *
 *
 *
 *
 */
template <class Op, int axis>
__global__ void kMatVect(Op op, float *A, float *x, float *B, int len, int size0)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len) {
		if (axis == 0) B[i] = op(A[i], x[i % size0]);
		if (axis == 1) B[i] = op(A[i], x[i / size0]);
	}
}

template <class Op>
int mat_vect(Op op, lua_State *L)
{
	THCudaTensor *A = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *B = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	int axis = luaL_checkint(L, 4) - 1;

	assert(axis == 0 || axis == 1);

	if (!is_rm(A) || !is_rm(B)) {
		luaL_error(L, "Matrix not in row major order");
	}

	if (THCudaTensor_nElement(A) != THCudaTensor_nElement(B)) {
		luaL_error(L, "Size mismatch");
	}

	int len = THCudaTensor_nElement(A);
	if (axis == 0) {
		if (A->size[1] != THCudaTensor_nElement(x)) {
			luaL_error(L, "Size mismatch");
		}
		kMatVect<Op, 0><<<(len - 1) / TB + 1, TB>>>(op, THCudaTensor_data(A), THCudaTensor_data(x), THCudaTensor_data(B), len, A->size[1]);
	} else if (axis == 1) {
		if (A->size[0] != THCudaTensor_nElement(x)) {
			luaL_error(L, "Size mismatch");
		}
		kMatVect<Op, 1><<<(len - 1) / TB + 1, TB>>>(op, THCudaTensor_data(A), THCudaTensor_data(x), THCudaTensor_data(B), len, A->size[1]);
	}

	checkCudaError(L);
	return 0;
}

int add_mat_vect(lua_State *L)
{
	return mat_vect(opPlus(), L);
}

int sub_mat_vect(lua_State *L)
{
	return mat_vect(opMinus(), L);
}

int mult_mat_vect(lua_State *L)
{
	return mat_vect(opMult(), L);
}

int div_mat_vect(lua_State *L)
{
	return mat_vect(opDiv(), L);
}

__global__ void kAdd(float *A, float *B, float *C, float alpha, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len) C[i] = A[i] + alpha * B[i];
}

/* C = A + alpha * B */
int add(lua_State *L)
{
	THCudaTensor *A = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *B = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *C = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	float alpha = luaL_optnumber(L, 4, 1.0);

	if (!is_rm(A) || !is_rm(B) || !is_rm(C)) {
		luaL_error(L, "Matrices not in row major order");
	}

	if (!(A->size[0] == B->size[0] && A->size[1] == B->size[1] && A->size[0] == C->size[0] && A->size[1] == C->size[1])) {
		luaL_error(L, "Size mismatch");
	}

	int len = THCudaTensor_nElement(A);
	kAdd<<<(len - 1) / TB + 1, TB>>>(THCudaTensor_data(A), THCudaTensor_data(B), THCudaTensor_data(C), alpha, len);
	checkCudaError(L);
	return 0;
}

/* What a crazy bug!
 *
 *
 *
 *
 *
 */
template <class Op>
__global__ void kReduce(Op op, float *A, float *x, int n, int axis)
{
	extern __shared__ float sdata[];

	int i = threadIdx.x;

	sdata[i] = op.base_value;
	if (i < n) {
		if (axis == 0) {
			sdata[i] = A[gridDim.x * threadIdx.x + blockIdx.x];
		} else if (axis == 1) {
			sdata[i] = A[threadIdx.x + n * blockIdx.x];
		}
	}
	__syncthreads();

	for (int s = blockDim.x >> 1; s > 0; s >>= 1) {
		if (i < s) {
			sdata[i] = op(sdata[i], sdata[i + s]);
		}
		__syncthreads();
	}

	if (i == 0) {
		x[blockIdx.x] = sdata[0];
	}
}

template <class Op>
int reduce(Op op, lua_State *L)
{
	int reduce_dim, other_dim;

	THCudaTensor *A = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	int axis = luaL_checkint(L, 3) - 1;

	if (!is_rm(A)) {
		luaL_error(L, "Matrix not in row major order");
	}

	if (axis != 0 && axis != 1) {
		luaL_error(L, "axis not in {0, 1}");
	}

	if (axis == 0) {
		reduce_dim = A->size[0];
		other_dim = A->size[1];
	} else if (axis == 1) {
		reduce_dim = A->size[1];
		other_dim = A->size[0];
	}

	assert(reduce_dim <= 1024);
	if (other_dim != THCudaTensor_nElement(x)) {
		luaL_error(L, "Size mismatch"); 
	}

	int threads = 1;
	while(threads < reduce_dim) {
		threads = threads << 1;
	}

	kReduce<Op><<<other_dim, threads, threads * sizeof(float)>>>(op, THCudaTensor_data(A), THCudaTensor_data(x), reduce_dim, axis);
	checkCudaError(L);
	return 0;
}

int sum(lua_State *L)
{
	return reduce(opPlus(), L);
}

int _max(lua_State *L)
{
	return reduce(opMax(), L);
}

static const struct luaL_Reg funcs[] = {
	{"add", add},
	{"add_mat_vect", add_mat_vect},
	{"cce", cce},
	{"div_mat_vect", div_mat_vect},
	{"exp", _exp},
	{"get_cols", get_cols},
	{"huber", huber},
	{"huber_deriv", huber_deriv},
	{"max", _max},
	{"mult_by_sigmoid_deriv", mult_by_sigmoid_deriv},
	{"mult_by_tanh_deriv", mult_by_tanh_deriv},
	{"mult_mat_vect", mult_mat_vect},
	{"set_cols", set_cols},
	{"shrink", shrink},
	{"sigmoid", sigmoid},
	{"sub_mat_vect", sub_mat_vect},
	{"sum", sum},
	{"tanh", tanh},

	{NULL, NULL}
};

extern "C" int luaopen_libjzt(lua_State *L) {
	luaL_openlib(L, "jzt", funcs, 0);
	return 1;
}
