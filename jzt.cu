#include "hip/hip_runtime.h"
extern "C" {
	#include "lua.h"
	#include "lualib.h"
	#include "lauxlib.h"
}

#include "luaT.h"
#include "THC.h"

#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

#include <stdio.h>
#include <assert.h>
#include "hipblas.h"

#define TB 128

/* operations */
struct opPlus {
	static const float base_value = 0.0;
	__device__ float operator()(float x, float y)
	{
		return x + y;
	}
};

struct opMinus {
	static const float base_value = 0.0;
	__device__ float operator()(float x, float y)
	{
		return x - y;
	}
};

struct opMult {
	static const float base_value = 1.0;
	__device__ float operator()(float x, float y)
	{
		return x * y;
	}
};

struct opDiv {
	static const float base_value = 1.0;
	__device__ float operator()(float x, float y)
	{
		return x / y;
	}
};

struct opSMul {
	float alpha;
	opSMul(float alpha_) : alpha(alpha_) {};
	__device__ float operator()(float x)
	{
		return alpha * x;
	}
};

struct opMax {
	static const float base_value = -2e38;
	__device__ float operator()(float x, float y)
	{
		return fmaxf(x, y);
	}
};

struct opClip {
	float low, high;
	opClip(float low_, float high_) : low(low_), high(high_) {};
	__device__ float operator()(float x)
	{
		return min(high, max(low, x));
	}
};

struct opExp {
	__device__ float operator()(float x)
	{
		return exp(x);
	}
};

struct opSigmoid {
	__device__ float operator()(float x)
	{
		return 1 / (1 + exp(-x));
	}
};

struct opSigmoidDeriv {
	__device__ float operator()(float x, float y)
	{
		return x * y * (1 - y);
	}
};

struct opTanh {
	__device__ float operator()(float x)
	{
		return tanh(x);
	}
};

struct opTanhDeriv {
	__device__ float operator()(float x, float y)
	{
		return x * (1 - y * y);
	}
};

struct opRelu {
	__device__ float operator()(float x)
	{
		return max(x, 0.f);
	}
};

struct opReluDeriv {
	__device__ float operator()(float x, float y)
	{
		return y > 0 ? x : 0;
	}
};

struct opHuber {
	float threshold;
	opHuber(float threshold_) : threshold(threshold_) {};
	__device__ float operator()(float x, float y) {
		float d = x - y;
		if (-threshold < d && d < threshold) {
			return 0.5 * d * d;
		} else {
			return threshold * (abs(d) - 0.5 * threshold);
		}
	}
};

struct opHuberDeriv {
	float threshold;
	opHuberDeriv(float threshold_) : threshold(threshold_) {};
	__device__ float operator()(float x, float y) {
		float d = x - y;
		if (-threshold < d && d < threshold) {
			return d;
		} else {
			return threshold * signbit(d);
		}
	}
};

struct opShrink {
	float threshold;
	opShrink(float threshold_) : threshold(threshold_) {};
	
	__device__ float operator()(float x) { 
		if (x - threshold > 0) {
			return x - threshold;
		} else if (x + threshold < 0) {
			return x + threshold;
		} else {
			return 0.0;
		}
	}
};

/* Is A in row major format? */
int is_rm(THCudaTensor *A)
{
	for (int i = 0; i < 4; i++) {
		if (A->nDimension == i + 1) return 1;
		if (A->stride[i] < A->stride[i + 1]) return 0;
	}
	assert(0);
	return 0;
}

void checkCudaError(lua_State *L) {
	hipError_t status = hipPeekAtLastError();
	if (status != hipSuccess) {
		luaL_error(L, hipGetErrorString(status));
	}
}

/* res[i] = A[inds[i]] */
__global__ void get_cols(float *A, int A_stride, float *inds, float *res, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len) {
		res[i] = A[i * A_stride + (int)inds[i] - 1];
	}
}

int get_cols(lua_State *L)
{
	THCudaTensor *A = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *inds = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *res = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	int len = THCudaTensor_nElement(inds);
	get_cols<<<(len - 1)  / TB + 1, TB>>>(THCudaTensor_data(A), A->stride[0], THCudaTensor_data(inds), THCudaTensor_data(res), len);
	return 0;
}


/* A[inds[i]] = val */
__global__ void set_cols(float *A, int A_stride, float *inds, float val, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len) {
		A[i * A_stride + (int)inds[i] - 1] = val;
	}
}

int set_cols(lua_State *L)
{
	THCudaTensor *A = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *inds = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	float val = luaL_checknumber(L, 3);
	int len = THCudaTensor_nElement(inds);
	set_cols<<<(len - 1)  / TB + 1, TB>>>(THCudaTensor_data(A), A->stride[0], THCudaTensor_data(inds), val, len);
	return 0;
}


template<class Op>
int transform1(Op op, lua_State *L)
{
	THCudaTensor *A = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *B = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	int lenA = THCudaTensor_nElement(A);
	int lenB = THCudaTensor_nElement(B);

	if (!is_rm(A) || !is_rm(B)) {
		luaL_error(L, "Matrices not in row major order");
	}

	if (lenA != lenB) {
		luaL_error(L, "Size mismatch");
	}

	thrust::device_ptr<float> pA(THCudaTensor_data(A));
	thrust::device_ptr<float> pB(THCudaTensor_data(B));
	thrust::transform(pA, pA + lenA, pB, op);
	return 0;
}

template<class Op>
int transform2(Op op, lua_State *L)
{
	THCudaTensor *A = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *B = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *C = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	int lenA = THCudaTensor_nElement(A);
	int lenB = THCudaTensor_nElement(B);
	int lenC = THCudaTensor_nElement(C);

	if (!is_rm(A) || !is_rm(B) || !is_rm(C)) {
		luaL_error(L, "Matrices not in roj major order");
	}

	if (lenA != lenB || lenA != lenC) {
		luaL_error(L, "Size mismatch");
	}

	thrust::device_ptr<float> pA(THCudaTensor_data(A));
	thrust::device_ptr<float> pB(THCudaTensor_data(B));
	thrust::device_ptr<float> pC(THCudaTensor_data(C));
	thrust::transform(pA, pA + lenA, pB, pC, op);
	return 0;
}

int huber(lua_State *L)
{
	float threshold = luaL_checknumber(L, 4);
	return transform2(opHuber(threshold), L);
}

int huber_deriv(lua_State *L)
{
	float threshold = luaL_checknumber(L, 4);
	return transform2(opHuberDeriv(threshold), L);
}

int shrink(lua_State *L)
{
	float threshold = luaL_checknumber(L, 3);
	return transform1(opShrink(threshold), L);
}

int sigmoid(lua_State *L)
{
	return transform1(opSigmoid(), L);
}

int mult_by_sigmoid_deriv(lua_State *L)
{
	return transform2(opSigmoidDeriv(), L);
}

int tanh(lua_State *L)
{
	return transform1(opTanh(), L);
}

int mult_by_tanh_deriv(lua_State *L)
{
	return transform2(opTanhDeriv(), L);
}

int relu(lua_State *L)
{
	return transform1(opRelu(), L);
}

int mult_by_relu_deriv(lua_State *L)
{
	return transform2(opReluDeriv(), L);
}

int clip(lua_State *L)
{
	float low = luaL_checknumber(L, 3);
	float high = luaL_checknumber(L, 4);
	return transform1(opClip(low, high), L);
}

int _exp(lua_State *L)
{
	return transform1(opExp(), L);
}

int smul(lua_State *L)
{
	float alpha = luaL_checknumber(L, 3);
	return transform1(opSMul(alpha), L);
}

/* What a crazy bug!
 *
 *
 *
 *
 *
 */
template <class Op, int axis>
__global__ void kMatVect(Op op, float *A, float *x, float *B, int len, int size0)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len) {
		if (axis == 0) B[i] = op(A[i], x[i % size0]);
		if (axis == 1) B[i] = op(A[i], x[i / size0]);
	}
}

template <class Op>
int mat_vect(Op op, lua_State *L)
{
	THCudaTensor *A = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *B = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	int axis = luaL_checkint(L, 4) - 1;

	assert(axis == 0 || axis == 1);

	if (!is_rm(A) || !is_rm(B)) {
		luaL_error(L, "Matrix not in row major order");
	}

	if (THCudaTensor_nElement(A) != THCudaTensor_nElement(B)) {
		luaL_error(L, "Size mismatch");
	}

	int len = THCudaTensor_nElement(A);
	if (axis == 0) {
		if (A->size[1] != THCudaTensor_nElement(x)) {
			luaL_error(L, "Size mismatch");
		}
		kMatVect<Op, 0><<<(len - 1) / TB + 1, TB>>>(op, THCudaTensor_data(A), THCudaTensor_data(x), THCudaTensor_data(B), len, A->size[1]);
	} else if (axis == 1) {
		if (A->size[0] != THCudaTensor_nElement(x)) {
			luaL_error(L, "Size mismatch");
		}
		kMatVect<Op, 1><<<(len - 1) / TB + 1, TB>>>(op, THCudaTensor_data(A), THCudaTensor_data(x), THCudaTensor_data(B), len, A->size[1]);
	}

	checkCudaError(L);
	return 0;
}

int add_mat_vect(lua_State *L)
{
	return mat_vect(opPlus(), L);
}

int sub_mat_vect(lua_State *L)
{
	return mat_vect(opMinus(), L);
}

int mult_mat_vect(lua_State *L)
{
	return mat_vect(opMult(), L);
}

int div_mat_vect(lua_State *L)
{
	return mat_vect(opDiv(), L);
}

__global__ void kAdd(float *A, float *B, float *C, float alpha, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len) C[i] = A[i] + alpha * B[i];
}

/* C = A + alpha * B */
int add(lua_State *L)
{
	THCudaTensor *A = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *B = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *C = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	float alpha = luaL_optnumber(L, 4, 1.0);

	if (!is_rm(A) || !is_rm(B) || !is_rm(C)) {
		luaL_error(L, "Matrices not in row major order");
	}

	if (!(A->size[0] == B->size[0] && A->size[1] == B->size[1] && A->size[0] == C->size[0] && A->size[1] == C->size[1])) {
		luaL_error(L, "Size mismatch");
	}

	int len = THCudaTensor_nElement(A);
	kAdd<<<(len - 1) / TB + 1, TB>>>(THCudaTensor_data(A), THCudaTensor_data(B), THCudaTensor_data(C), alpha, len);
	checkCudaError(L);
	return 0;
}

/* What a crazy bug!
 *
 *
 *
 *
 *
 */
template <class Op>
__global__ void kReduce(Op op, float *A, float *x, int n, int axis)
{
	extern __shared__ float sdata[];

	int i = threadIdx.x;

	sdata[i] = op.base_value;
	if (i < n) {
		if (axis == 0) {
			sdata[i] = A[gridDim.x * threadIdx.x + blockIdx.x];
		} else if (axis == 1) {
			sdata[i] = A[threadIdx.x + n * blockIdx.x];
		}
	}
	__syncthreads();

	for (int s = blockDim.x >> 1; s > 0; s >>= 1) {
		if (i < s) {
			sdata[i] = op(sdata[i], sdata[i + s]);
		}
		__syncthreads();
	}

	if (i == 0) {
		x[blockIdx.x] = sdata[0];
	}
}

template <class Op>
int reduce(Op op, lua_State *L)
{
	int reduce_dim, other_dim;

	THCudaTensor *A = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	int axis = luaL_checkint(L, 3) - 1;

	if (!is_rm(A)) {
		luaL_error(L, "Matrix not in row major order");
	}

	if (axis != 0 && axis != 1) {
		luaL_error(L, "axis not in {0, 1}");
	}

	if (axis == 0) {
		reduce_dim = A->size[0];
		other_dim = A->size[1];
	} else if (axis == 1) {
		reduce_dim = A->size[1];
		other_dim = A->size[0];
	}

	assert(reduce_dim <= 1024);
	if (other_dim != THCudaTensor_nElement(x)) {
		luaL_error(L, "Size mismatch"); 
	}

	int threads = 1;
	while(threads < reduce_dim) {
		threads = threads << 1;
	}

	kReduce<Op><<<other_dim, threads, threads * sizeof(float)>>>(op, THCudaTensor_data(A), THCudaTensor_data(x), reduce_dim, axis);
	checkCudaError(L);
	return 0;
}

int sum(lua_State *L)
{
	return reduce(opPlus(), L);
}

int _max(lua_State *L)
{
	return reduce(opMax(), L);
}

__global__ void kShrink2(float *x1, float *x2, float l, float g, int len)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) {
		float u, v;
		// This must be painfully slow because of branching
		u = -l -g +x1[i]; v = -l +g +x2[i]; if (u > v && v > 0) goto end;
		u = -l -g +x1[i]; v = +l +g +x2[i]; if (u > 0 && 0 > v) goto end;
		u = -l +g +x1[i]; v = -l -g +x2[i]; if (v > u && u > 0) goto end;
		u = +l +g +x1[i]; v = -l -g +x2[i]; if (v > 0 && 0 > u) goto end;
		u = +l -g +x1[i]; v = +l +g +x2[i]; if (0 > u && u > v) goto end;
		u = +l +g +x1[i]; v = +l -g +x2[i]; if (0 > v && v > u) goto end;
end:
		x1[i] = u;
		x2[i] = v;
    }
}

int shrink2(lua_State *L)
{
	THCudaTensor *x1 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x2 = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	float lambda = luaL_checknumber(L, 3);
	float gamma = luaL_checknumber(L, 4);

	int x1_size = THCudaTensor_nElement(x1);
	int x2_size = THCudaTensor_nElement(x2);
	
	if (!is_rm(x1) && !is_rm(x2)) {
		luaL_error(L, "Matrix not in row major order");
	}

	if (x1_size != x2_size) {
		luaL_error(L, "Size mismatch");
	}

	kShrink2<<<(x1_size - 1)  / TB + 1, TB>>>(THCudaTensor_data(x1), THCudaTensor_data(x2), lambda, gamma, x1_size);
	checkCudaError(L);
	return 0;
}

__global__ void spatial_argmax_kernel(float *input, float *output, int size, int size1, int size23)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int dim23 = id % size23;
		int dim0 = id / size23;

		int argmax;
		float max = -2e38;
		for (int i = 0; i < size1; i++) {
			float val = input[(dim0 * size1 + i) * size23 + dim23];
			if (val > max) {
				max = val;
				argmax = i;
			}
		}
		output[id] = argmax + 1;
	}
}

int spatial_argmax(lua_State *L)
{
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

	if (!is_rm(input) && !is_rm(output)) {
		luaL_error(L, "Matrix not in row major order");
	}

	if (input->nDimension != 4 || output->nDimension != 4) {
		luaL_error(L, "Number of dimensions has to be 4");
	}

	if (THCudaTensor_size(input, 0) != THCudaTensor_size(output, 0) ||
	  THCudaTensor_size(output, 1) != 1 ||
	  THCudaTensor_size(input, 2) != THCudaTensor_size(output, 2) ||
	  THCudaTensor_size(input, 3) != THCudaTensor_size(output, 3)) {
		luaL_error(L, "Size mismatch");
	}

	int size = THCudaTensor_nElement(output);
	spatial_argmax_kernel<<<(size - 1) / TB + 1, TB>>>(
		THCudaTensor_data(input), 
		THCudaTensor_data(output), 
		size,
		THCudaTensor_size(input, 1),
		THCudaTensor_size(input, 2) * THCudaTensor_size(output, 3));
	checkCudaError(L);
	return 0;
}

__global__ void sc1_updateOutput_kernel(float *input, float *weight, int transpose_weight, float *output, int img_size, int num_input, int num_output)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int batch = blockIdx.y;
	float input_reg[32];

	__shared__ float weight_s[32 * 32];
	for (int i = threadIdx.x; i < num_input * num_output; i += blockDim.x) {
		if (transpose_weight) {
			weight_s[(i % num_output) * num_input + (i / num_output)] = weight[i];
		} else {
			weight_s[i] = weight[i];
		}
	}
	__syncthreads();

	if (id < img_size) { 
		for (int j = 0; j < num_input; j++) {
			input_reg[j] = input[(batch * num_input + j) * img_size + id];
		}

		for (int i = 0; i < num_output; i++) {
			float s = 0;
			for (int j = 0; j < num_input; j++) {
				s += input_reg[j] * weight_s[i * num_input + j];
			}
			output[(batch * num_output + i) * img_size + id] = s;
		}
	}
}

int sc1_updateOutput(lua_State *L)
{
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *weight = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	int transpose_weight = luaL_checkinteger(L, 3);
	THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");

	int batch_size = THCudaTensor_size(input, 0);
	int img_size = THCudaTensor_size(input, 2) * THCudaTensor_size(input, 3);

	int num_input, num_output;
	if (transpose_weight) {
		num_input = THCudaTensor_size(weight, 0);
		num_output = THCudaTensor_size(weight, 1);
	} else {
		num_input = THCudaTensor_size(weight, 1);
		num_output = THCudaTensor_size(weight, 0);
	}

	if (!is_rm(input) || !is_rm(weight) || !is_rm(output)) {
		luaL_error(L, "Matrix not in row major order");
	}

	assert(num_input <= 32 && num_input * num_output <= 32 * 32);

	dim3 grid((img_size - 1) / TB + 1, batch_size);
	sc1_updateOutput_kernel<<<grid, TB>>>(
		THCudaTensor_data(input), 
		THCudaTensor_data(weight), 
		transpose_weight,
		THCudaTensor_data(output), 
		img_size, num_input, num_output);

	checkCudaError(L);
	return 0;
}

__global__ void sc1_accGradParameters_kernel(float *input, float *grad_output, float *grad, int batch_size, int img_size, int num_input, int num_output)
{
	__shared__ float input_s[32 * 32];
	__shared__ float grad_output_s[32 * 32];

	for (int i = threadIdx.y * blockDim.x + threadIdx.x; i < num_input * batch_size; i += blockDim.x * blockDim.y) {
		input_s[i] = input[i * img_size + blockIdx.x];
	}

	for (int i = threadIdx.y * blockDim.x + threadIdx.x; i < num_output * batch_size; i += blockDim.x * blockDim.y) {
		grad_output_s[i] = grad_output[i * img_size + blockIdx.x];
	}

	__syncthreads();
	
	float s = 0;
	for (int k = 0; k < batch_size; k++) {
		s += grad_output_s[k * num_output + threadIdx.x] * input_s[k * num_input + threadIdx.y];
	}
	
	atomicAdd(grad + threadIdx.x * num_input + threadIdx.y, s);
}

int sc1_accGradParameters(lua_State *L)
{
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *grad_output = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *grad = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");

	int batch_size = THCudaTensor_size(input, 0);
	int img_size = THCudaTensor_size(input, 2) * THCudaTensor_size(input, 3);
	int num_input = THCudaTensor_size(input, 1);
	int num_output = THCudaTensor_size(grad_output, 1);

	if (!is_rm(input) || !is_rm(grad_output) || !is_rm(grad)) {
		luaL_error(L, "Matrix not in row major order");
	}

	assert(num_input <= 32 && batch_size <= 32 && num_input * num_output <= 32 * 32);
	dim3 block(num_output, num_input);
	sc1_accGradParameters_kernel<<<img_size, block>>>(THCudaTensor_data(input), THCudaTensor_data(grad_output), THCudaTensor_data(grad), batch_size, img_size, num_input, num_output);

	checkCudaError(L);
	return 0;
}

__global__ void add_bias4_kernel(float *input, float *bias, int input_size, int bias_size, int img_size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < input_size) {
		input[id] += bias[(id / img_size) % bias_size];
	}
}

int add_bias4(lua_State *L)
{
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *bias = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

	assert(THCudaTensor_size(input, 1) == THCudaTensor_nElement(bias));
	assert(THCudaTensor_size(input, 1) <= 32);

	if (!is_rm(input) || !is_rm(bias)) {
		luaL_error(L, "Matrix not in row major order");
	}

	add_bias4_kernel<<<(THCudaTensor_nElement(input) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(input), 
		THCudaTensor_data(bias), 
		THCudaTensor_nElement(input),
		THCudaTensor_size(input, 1),
		THCudaTensor_size(input, 2) * THCudaTensor_size(input, 3));
	checkCudaError(L);
	return 0;
}

__global__ void stereoJoin_updateOutput_kernel(float *left, float *right, float *output, int size_out, int size1_out, int size2, int size3, int size1_in)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size_out) {
		int dim3 = id % size3;
		id /= size3;
		int dim2 = id % size2;
		id /= size2;
		int dim1 = id % size1_out;
		int dim0 = id / size1_out;

		float d;
		if (dim3 >= dim1) {	
			d = 0;
			for (int i = 0; i < size1_in; i++) {
				float l = left[((dim0 * size1_in + i) * size2 + dim2) * size3 + dim3];
				float r = right[((dim0 * size1_in + i) * size2 + dim2) * size3 + dim3 - dim1];
				float dd = l - r;
				d += dd * dd;	
			}
		} else {
			d = 2e38;
		}
		output[((dim0 * size1_out + dim1) * size2 + dim2) * size3 + dim3] = -d;
	}
}

int stereoJoin_updateOutput(lua_State *L)
{
	THCudaTensor *left = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *right = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	
	if (!is_rm(left) || !is_rm(right) || !is_rm(output)) {
		luaL_error(L, "Matrix not in row major order");
	}

	stereoJoin_updateOutput_kernel<<<(THCudaTensor_nElement(output) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(left),
		THCudaTensor_data(right),
		THCudaTensor_data(output),
		THCudaTensor_nElement(output),
		THCudaTensor_size(output, 1),
		THCudaTensor_size(output, 2),
		THCudaTensor_size(output, 3),
		THCudaTensor_size(left, 1));

	checkCudaError(L);
	return 0;
}

__global__ void stereoJoin_updateGradInput_kernel(float *left, float *right, float *gradOutput, float *leftGrad, float *rightGrad, int size_out, int size1_out, int size2, int size3, int size1_in)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size_out) {
		int dim3 = id % size3;
		id /= size3;
		int dim2 = id % size2;
		id /= size2;
		int dim1 = id % size1_out;
		int dim0 = id / size1_out;

		/* leftGrad */ 
		float d = 0.;
		float l = left[((dim0 * size1_out + dim1) * size2 + dim2) * size3 + dim3];
		for (int i = 0; i < size1_in && dim3 - i >= 0; i++) {
			float r = right[((dim0 * size1_out + dim1) * size2 + dim2) * size3 + dim3 - i];
			float g = gradOutput[((dim0 * size1_in + i) * size2 + dim2) * size3 + dim3];
			d += 2 * (l - r) * g;
		}
		leftGrad[((dim0 * size1_out + dim1) * size2 + dim2) * size3 + dim3] = -d;

		/* rightGrad */
		d = 0.;
		float r = right[((dim0 * size1_out + dim1) * size2 + dim2) * size3 + dim3];
		for (int i = 0; i < size1_in && dim3 + i < size3; i++) {
			float l = left[((dim0 * size1_out + dim1) * size2 + dim2) * size3 + dim3 + i];
			float g = gradOutput[((dim0 * size1_in + i) * size2 + dim2) * size3 + dim3 + i];
			d += 2 * (r - l) * g;
		}
		rightGrad[((dim0 * size1_out + dim1) * size2 + dim2) * size3 + dim3] = -d;
	}
}

int stereoJoin_updateGradInput(lua_State *L)
{
	THCudaTensor *left = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *right = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *leftGrad = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	THCudaTensor *rightGrad = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");

	if (!is_rm(left) || !is_rm(right) || !is_rm(leftGrad) || !is_rm(rightGrad)) {
		luaL_error(L, "Matrix not in row major order");
	}

	stereoJoin_updateGradInput_kernel<<<(THCudaTensor_nElement(left) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(left),
		THCudaTensor_data(right),
		THCudaTensor_data(gradOutput),
		THCudaTensor_data(leftGrad),
		THCudaTensor_data(rightGrad),
		THCudaTensor_nElement(left),
		THCudaTensor_size(left, 1),
		THCudaTensor_size(left, 2),
		THCudaTensor_size(left, 3),
		THCudaTensor_size(gradOutput, 1));
	checkCudaError(L);
	return 0;
}

/* CPU implementation */
int depth2disp(lua_State *L)
{
	THFloatTensor *input = (THFloatTensor*)luaT_checkudata(L, 1, "torch.FloatTensor");
	float c = luaL_checknumber(L, 2);

	float *input_p = THFloatTensor_data(input);
	int size = THFloatTensor_nElement(input);

	for (int i = 0; i < size; i++) {
		if (input_p[i] != 0.0) {
			input_p[i] = c / input_p[i];
		}
	}

	return 0;
}

/* CPU implementation */
int grey2jet(lua_State *L)
{
	THDoubleTensor *grey_img = (THDoubleTensor*)luaT_checkudata(L, 1, "torch.DoubleTensor");
	THDoubleTensor *col_img = (THDoubleTensor*)luaT_checkudata(L, 2, "torch.DoubleTensor");

	assert(grey_img->nDimension == 2);
	if (3 * THDoubleTensor_nElement(grey_img) != THDoubleTensor_nElement(col_img)) {
		luaL_error(L, "Size mismatch");
	}

	int height = THDoubleTensor_size(grey_img, 0);
	int width = THDoubleTensor_size(grey_img, 1);

	double *gray_data = THDoubleTensor_data(grey_img);
	double *col_data = THDoubleTensor_data(col_img);

	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			double val = gray_data[i * width + j] * 4;
			double r = 0, g = 0, b = 0;

			if (-0.1 <= val && val < 0.5) {
				r = 0;
				g = 0;
				b = 0.5 + val;
			} else if (0.5 <= val && val < 1.5) {
				r = 0;
				g = val - 0.5;
				b = 1;
			} else if (1.5 <= val && val < 2.5) {
				r = val - 1.5;
				g = 1;
				b = 1 - (val - 1.5);
			} else if (2.5 <= val && val < 3.5) {
				r = 1;
				g = 1 - (val - 2.5);
				b = 0;
			} else if (3.5 <= val && val <= 4.1) {
				r = 1 - (val - 3.5);
				g = 0;
				b = 0;
			} else {
				printf("val = %f\n", val);
				assert(0);
			}

			col_data[(0 * height + i) * width + j] = r;
			col_data[(1 * height + i) * width + j] = g;
			col_data[(2 * height + i) * width + j] = b;
		}
	}
	return 0;
}

__global__ void L2Pooling_updateOutput_kernel(float *input, float *output, int ksize, int stride, int size, int width, int height, int pooled_width, int pooled_height)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int dim01 = id;
	const int col_out = dim01 % pooled_width;
	dim01 /= pooled_width;
	const int row_out = dim01 % pooled_height;
	dim01 /= pooled_height;

	if (id < size) {
		const int row_in = row_out * stride;
		const int col_in = col_out * stride;
		const int offset_in = dim01 * width * height;
		float val = 0;
		for (int i = 0; i < ksize; i++) {
			for (int j = 0; j < ksize; j++) {
				float d = input[offset_in + (row_in + i) * width + (col_in + j)];
				val += d * d;
			}
		}
		output[id] = sqrtf(val);
	}
}

int L2Pooling_updateOutput(lua_State *L) 
{
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	int ksize = luaL_checkinteger(L, 3);
	int stride = luaL_checkinteger(L, 4);

	int batch_size = THCudaTensor_size(input, 0);
	int img_size = THCudaTensor_size(input, 2) * THCudaTensor_size(input, 3);

	if (!is_rm(input) || !is_rm(output)) {
		luaL_error(L, "Matrix not in row major order");
	}

	const int height = THCudaTensor_size(input, 2);
	const int width = THCudaTensor_size(input, 3);

	const int pooled_height = floor((float)(height - ksize) / stride) + 1;
	const int pooled_width = floor((float)(width - ksize) / stride) + 1;

	assert(THCudaTensor_size(output, 2) == pooled_height);
	assert(THCudaTensor_size(output, 3) == pooled_width);

	L2Pooling_updateOutput_kernel<<<(THCudaTensor_nElement(output) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(input), 
		THCudaTensor_data(output), 
		ksize, stride, THCudaTensor_nElement(output), width, height, pooled_width, pooled_height);
	
	return 0;
}

__global__ void L2Pooling_updateGradInput_kernel(float *input, float *output, float *gradInput, float *gradOutput, int ksize, int stride, int size, int width, int height, int pooled_width, int pooled_height)
{
	int output_id = blockIdx.x * blockDim.x + threadIdx.x;
	int dim01 = output_id;
	const int col_out = dim01 % pooled_width;
	dim01 /= pooled_width;
	const int row_out = dim01 % pooled_height;
	dim01 /= pooled_height;

	if (output_id < size) {
		const int row_in = row_out * stride;
		const int col_in = col_out * stride;
		const int offset_in = dim01 * width * height;
		for (int i = 0; i < ksize; i++) {
			for (int j = 0; j < ksize; j++) {
				const int input_id = offset_in + (row_in + i) * width + (col_in + j);
				atomicAdd(gradInput + input_id, input[input_id] * gradOutput[output_id] / output[output_id]);
			}
		}
	}
}

int L2Pooling_updateGradInput(lua_State *L)
{
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *gradInput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	int ksize = luaL_checkinteger(L, 5);
	int stride = luaL_checkinteger(L, 6);

	L2Pooling_updateGradInput_kernel<<<(THCudaTensor_nElement(output) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(input),
		THCudaTensor_data(output),
		THCudaTensor_data(gradInput),
		THCudaTensor_data(gradOutput),
		ksize, stride,
		THCudaTensor_nElement(output),
		THCudaTensor_size(input, 3),
		THCudaTensor_size(input, 2),
		THCudaTensor_size(output, 3),
		THCudaTensor_size(output, 2));
	return 0;
}


__global__ void ConvSplit_updateOutput_kernel(float *input, float *output, int output_size, int win_size, int overlap, int ncol, int nrow, int width, int height)
{
	int output_id = blockIdx.x * blockDim.x + threadIdx.x;

	if (output_id < output_size) {
		int id = output_id;
		const int x = id % win_size;
		id /= win_size;
		const int y = id % win_size;
		id /= win_size;
		const int col = id % ncol;
		id /= ncol;
		const int row = id % nrow;

		const int ii = row * (win_size - 2 * overlap) + y;
		const int jj = col * (win_size - 2 * overlap) + x;
		if (ii < height && jj < width) {
			output[output_id] = input[ii * width + jj];
		} else {
			output[output_id] = 0;
		}
	}
}

int ConvSplit_updateOutput(lua_State *L)
{
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	const int win_size = luaL_checkinteger(L, 3);
	const int overlap = luaL_checkinteger(L, 4);
	const int nrow = luaL_checkinteger(L, 5);
	const int ncol = luaL_checkinteger(L, 6);

	const int height = THCudaTensor_size(input, 2);
	const int width = THCudaTensor_size(input, 3);

	ConvSplit_updateOutput_kernel<<<(THCudaTensor_nElement(output) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(input),
		THCudaTensor_data(output),
		THCudaTensor_nElement(output),
		win_size, overlap, ncol, nrow, width, height);
	return 0;
}

__global__ void ConvJoin_updateOutput_kernel(float *input, float *output, int output_size, int nimg, int win_size, int width, int height, int ncol)
{
	int output_id = blockIdx.x * blockDim.x + threadIdx.x;

	if (output_id < output_size) {
		int id = output_id;
		const int x = id % width;
		id /= width;
		const int y = id % height;
		id /= height;
		const int img = id;

		const int col = x / win_size;
		const int row = y / win_size;
		const int xx = x % win_size;
		const int yy = y % win_size;
		
		output[output_id] = input[(((row * ncol + col) * nimg + img) * win_size + yy) * win_size + xx];
	}
}

int ConvJoin_updateOutput(lua_State *L)
{
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

	const int height = THCudaTensor_size(output, 2);
	const int width = THCudaTensor_size(output, 3);
	const int win_size = THCudaTensor_size(input, 2);
	assert(win_size == THCudaTensor_size(input, 3));
	const int ncol = ceil((double)width / win_size);
	assert(ncol * ncol == THCudaTensor_size(input, 0));

	ConvJoin_updateOutput_kernel<<<(THCudaTensor_nElement(output) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(input),
		THCudaTensor_data(output),
		THCudaTensor_nElement(output),
		THCudaTensor_size(input, 1),
		win_size, width, height, ncol);
	return 0;
}

static const struct luaL_Reg funcs[] = {
	{"add", add},
	{"add_mat_vect", add_mat_vect},
	{"clip", clip},
	{"div_mat_vect", div_mat_vect},
	{"exp", _exp},
	{"get_cols", get_cols},
	{"huber", huber},
	{"huber_deriv", huber_deriv},
	{"max", _max},
	{"mult_by_relu_deriv", mult_by_relu_deriv},
	{"mult_by_sigmoid_deriv", mult_by_sigmoid_deriv},
	{"mult_by_tanh_deriv", mult_by_tanh_deriv},
	{"mult_mat_vect", mult_mat_vect},
	{"relu", relu},
	{"set_cols", set_cols},
	{"shrink", shrink},
	{"sigmoid", sigmoid},
	{"smul", smul},
	{"spatial_argmax", spatial_argmax},
	{"sub_mat_vect", sub_mat_vect},
	{"sum", sum},
	{"tanh", tanh},

	{"sc1_updateOutput", sc1_updateOutput},
	{"sc1_accGradParameters", sc1_accGradParameters},
	{"add_bias4", add_bias4},

	{"L2Pooling_updateOutput", L2Pooling_updateOutput},
	{"L2Pooling_updateGradInput", L2Pooling_updateGradInput},

	{"stereoJoin_updateOutput", stereoJoin_updateOutput},
	{"stereoJoin_updateGradInput", stereoJoin_updateGradInput},

	{"ConvSplit_updateOutput", ConvSplit_updateOutput},
	{"ConvJoin_updateOutput", ConvJoin_updateOutput},

	{"depth2disp", depth2disp},
	{"grey2jet", grey2jet},

	{NULL, NULL}
};

void cunn_SpatialLogSoftMax_init(lua_State *L);

extern "C" int luaopen_libjzt(lua_State *L) {
	luaL_openlib(L, "jzt", funcs, 0);
	cunn_SpatialLogSoftMax_init(L);
	return 1;
}
